#include "hip/hip_runtime.h"
// cuda_kernels.cu
#include "cuda_kernels.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAddKernel(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

void vectorAdd(const float* A, const float* B, float* C, int N) 
{
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Allocate memory on GPU
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy vectors from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "blocksPerGrid = " << blocksPerGrid << std::endl;
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();
    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free memory on GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
